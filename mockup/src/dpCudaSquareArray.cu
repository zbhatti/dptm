#include "hip/hip_runtime.h"
#include "dpCudaSquareArray.hpp"
#include "errorCheck.hpp"
#define cudaErrChk(ans) { cudaAssert((ans), __FILE__, __LINE__); }
#define BEGIN hipEventRecord(begin, 0);
#define END hipEventRecord(end, 0); hipEventSynchronize(end); hipEventElapsedTime(&delTime, begin, end);

// Kernel that executes on the CUDA device
__global__ void squareArray(float *Ain_d, float *Aout_d, int N){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) 
		Aout_d[idx] = Ain_d[idx] * Ain_d[idx];
}

//notice unused parameters for CUDA kernel:
dpCudaSquareArray::dpCudaSquareArray(cl_context ctx, cl_command_queue q){

	workDimension = ONE_D;
	//name is same as cl alternative allowing the analysis script to later figure 
	//out this measurement was from a cuda kernel by inspecting the platform id from dpClient
	name = "SquareArray";

	hipEventCreate(&begin);
	hipEventCreate(&end);
	
	hipGetDevice(&device);
	hipGetDeviceProperties(&props, device);
}

void dpCudaSquareArray::setup(int dataMB, int xLocal, int yLocal, int zLocal){

	localSize[0] = localSize[1] = localSize[2] = 1;
	
	Asize = 1048576*dataMB/sizeof(float);
	MB = Asize * sizeof(float) / 1048576;
	
}

void dpCudaSquareArray::init(){
	//allocate local memory for original array
	Ain = (float*) malloc(Asize*sizeof(float));
	Aout = (float*) malloc(Asize*sizeof(float));
	if (!Aout || !Ain)
		fprintf(stderr,"error in malloc\n");
	
	generateArray(Ain, Asize);
	dataParameters.push_back(Asize);
	dataNames.push_back("nElements");

}

void dpCudaSquareArray::memoryCopyOut(){
	BEGIN
	cudaErrChk( hipMalloc((void **) &Ain_d, Asize*sizeof(float) ));
	cudaErrChk( hipMalloc((void **) &Aout_d, Asize*sizeof(float) ));
	cudaErrChk( hipMemcpy(Ain_d, Ain, Asize*sizeof(float), hipMemcpyHostToDevice) );
	END
	//printf("%0.3f,",delTime);
}

void dpCudaSquareArray::plan(){
	BEGIN
	blockSize = props.maxThreadsPerBlock;
	lastBlock = 0;
	nBlocks = Asize/blockSize; //nblocks = ceil(Asize/blockSize)
	if (Asize%blockSize != 0)
		nBlocks++;
	if (nBlocks > 65535)
		nBlocks = 65535;
	nKernels = nBlocks / 65535;
	if (nKernels == 0){
		lastBlock = nBlocks; //run normally
	}
	else 
		lastBlock = nBlocks % 65535; //run repeated
	END
	
}

int dpCudaSquareArray::execute(){
	hipError_t err;
	BEGIN
	for (int i = 0; i < nKernels; i++)
		squareArray <<< nBlocks, blockSize >>> (A_d + (i*blockSize*nBlocks*sizeof(float)), B_d + (i*blockSize*nBlocks*sizeof(float)), Asize);
	if (lastBlock != 0)
		squareArray <<<lastBlock, blockSize >>> (A_d + (nKernels*blockSize*nBlocks*sizeof(float)), B_d + (nKernels*blockSize*nBlocks*sizeof(float)), Asize);
	err = hipPeekAtLastError() ;
	cudaErrChk(err);
	cudaErrChk(hipDeviceSynchronize());
	END
	if(err!=hipSuccess)
		return -1;
	return 0;
}

void dpCudaSquareArray::memoryCopyIn(){
	BEGIN
	cudaErrChk(hipMemcpy(Aout, Aout_d, Asize*sizeof(float), hipMemcpyDeviceToHost));
	END
	//printf("%0.3f,\n",delTime);
}

void dpCudaSquareArray::cleanUp(){
	hipFree(Ain_d);
	hipFree(Aout_d);
	free(Ain);
	free(Aout);
}

void dpCudaSquareArray::generateArray(float *A, int N){
	int i;
	srand(time(NULL));
	for (i=0; i < N; i++){
		A[i]=rand() / (RAND_MAX/99999.9 + 1);
	}
}


/*
#include <stddef.h>
#include <sys/types.h>
#include <unistd.h>
#include <stdlib.h>
#include <sys/un.h>
#include <math.h>
#include <errno.h>
#include <string.h>


*/



