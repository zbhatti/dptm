#include "hip/hip_runtime.h"
#include "dpCudaEmpty.hpp"
#include "errorCheck.hpp"
#define cudaErrChk(ans) { cudaAssert((ans), __FILE__, __LINE__); }
#define BEGIN hipEventRecord(begin, 0);
#define END hipEventRecord(end, 0); hipEventSynchronize(end); hipEventElapsedTime(&delTime, begin, end);

// Kernel that executes on the CUDA device
__global__ void empty(){}

//notice unused parameters for CUDA kernel:
dpCudaEmpty::dpCudaEmpty(cl_context ctx, cl_command_queue q){
	workDimension = ONE_D;
	//name is same as cl alternative allowing the analysis script to later figure 
	//out this measurement was from a cuda kernel by inspecting the platform id from dpClient
	name = "Empty";

	hipEventCreate(&begin);
	hipEventCreate(&end);
	
	hipGetDevice(&device);
	hipGetDeviceProperties(&props, device);
}

void dpCudaEmpty::setup(int dataMB, int xLocal, int yLocal, int zLocal){
	localSize[0] = localSize[1] = localSize[2] = 1;
	MB = dataMB;
}

void dpCudaEmpty::init(){}

void dpCudaEmpty::memoryCopyOut(){}

void dpCudaEmpty::plan(){
	BEGIN
	blockSize = props.maxThreadsPerBlock;
	nBlocks = 1024;
	END
}

int dpCudaEmpty::execute(){
	hipError_t err;
	BEGIN
	empty <<< nBlocks, blockSize >>>();
	err = hipPeekAtLastError() ;
	cudaErrChk(err);
	cudaErrChk(hipDeviceSynchronize());
	END
	//printf("%0.3f,",delTime);
	if(err!=hipSuccess)
		return -1;
	return 0;
}

void dpCudaEmpty::memoryCopyIn(){}

void dpCudaEmpty::cleanUp(){}
