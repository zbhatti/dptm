#include <sys/types.h>
#include <sys/socket.h>
#include <unistd.h>//using for sleep function and others
#include <stdlib.h>
#include <sys/un.h>
#include <stdio.h>
#include <sys/types.h>//shared memory
#include <sys/ipc.h>//shared memory
#include <sys/shm.h>//shared memory
#include <hip/hip_runtime.h>//cuda functions
#include <hip/hip_runtime.h>//cuda functions
#include <hipfft/hipfft.h>
#include <sys/time.h>//time funcion for clocking
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
#include "sharedfunctions.h"
#define START gettimeofday(&sT, NULL);
#define FINISH gettimeofday(&fT, NULL);

//code from stackexchange to help print cuda return messages
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true){
	if (code != hipSuccess) 
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

//used by wrapper and clients to make original socket connections
int clientSocketConnect(char *argv[]){
	int sockfd;
	struct sockaddr_un serv_addr;

	bzero((char*)&serv_addr, sizeof(serv_addr));
	serv_addr.sun_family = AF_UNIX;
	serv_addr.sun_len = sizeof(serv_addr);
	strcpy(serv_addr.sun_path, argv[1]);
	if ((sockfd = socket(AF_UNIX, SOCK_STREAM,0)) < 0)
		return -1;
	if (connect(sockfd, (struct sockaddr *) &serv_addr, sizeof(struct sockaddr_un)) < 0) 
		return -1;
	return sockfd;
}

int main(int argc, char *argv[]){

int sockfd, r, v, Asize;
hipfftComplex *massA_d;
bin k1;
int n[1] = {0};
struct timeval start, finish, sT, fT;

int num_devices, device;

hipGetDeviceCount(&num_devices);

fprintf(stderr,"Number of devices found: %d\n",num_devices);

if (num_devices > 1) {

	int max_multiprocessors = 0, max_device = 0;
	for (device = 0; device < num_devices; device++) {
		hipDeviceProp_t properties;
		hipGetDeviceProperties(&properties, device);

		fprintf(stderr,"Device %d has %d multiProcessors. \n",device, properties.multiProcessorCount);

		if (max_multiprocessors < properties.multiProcessorCount) {
			max_multiprocessors = properties.multiProcessorCount;
			max_device = device;
		}
	}

	//optionally select GPU
	if (argc>5)
		max_device=atoi(argv[5]);

	fprintf(stderr,"Selected Device %d.\n",max_device);
	hipSetDevice(max_device);
}



//cufft library variables
hipfftHandle plancufft;

//connect to DPTM
sockfd = clientSocketConnect(argv);
if (sockfd < 0)
	return -1;	
printf("BinCondition,NumberRequests,SizePerClient,ShmemAtt,Shared2GPU,CufftPlan,CufftExec,Gpu2Shared,ShmemDet,TotalTimeSpent\n");
//MAIN LOOP:	
while(1){

	recv(sockfd, &k1, sizeof(bin), 0); //sets sockfd write in select
	printf("%s,%d,%d,", k1.bincondition, k1.nreqs, k1.Asize);
	gettimeofday(&start, NULL);		

	Asize = k1.Asize;
	n[0] = Asize;


	//attach to shared memories
	START
	for(r = 0; r < k1.nreqs; r++){
		k1.shmid_ptrs[r] = (hipfftComplex*) shmat(k1.shmid[r], NULL, 0);
		if( k1.shmid_ptrs[r] == (void*) -1){
			printf("shmat failed");
			return -1;
		}
	}
	FINISH
	printf("%ld,", timediff(sT, fT));


	//copy the sequences to device
	START
	gpuErrchk(hipMalloc(&massA_d, Asize * k1.nreqs * sizeof(hipfftComplex) ));
	for (r = 0, v = 0; r < k1.nreqs; r++){
		gpuErrchk(hipMemcpy(&massA_d[v], k1.shmid_ptrs[r], sizeof(hipfftComplex) * Asize, hipMemcpyHostToDevice));
		v = v + Asize;
	}
	FINISH
	printf("%ld,", timediff(sT, fT));

	//create plan for cufft
	START
	if (hipfftPlanMany(&plancufft, 1, n, NULL, 1, 0, NULL, 1, 0, HIPFFT_C2C, k1.nreqs) != HIPFFT_SUCCESS){
		//cufftplan failed (should probably fit this into a function)
		gpuErrchk(hipFree(massA_d));
		for(r = 0; r < k1.nreqs; r++){
			shmdt((void*) k1.shmid_ptrs[r]);
		}
		hipfftDestroy(plancufft);
		gettimeofday(&finish, NULL);
		send(sockfd, &k1, sizeof(bin), 0);
		continue;
	}
	FINISH
	printf("%ld,", timediff(sT, fT) );

	//execute cufft
	START		
	if (hipfftExecC2C(plancufft, massA_d, massA_d, HIPFFT_FORWARD)!= HIPFFT_SUCCESS){
		//cufftexec failed (should probably fit this into a function)
		gpuErrchk(hipFree(massA_d));
		for(r = 0; r < k1.nreqs; r++){
			shmdt((void*) k1.shmid_ptrs[r]);
		}
		hipfftDestroy(plancufft);
		gettimeofday(&finish, NULL);
		send(sockfd, &k1, sizeof(bin), 0);
		continue;
	}
	FINISH
	printf("%ld,", timediff(sT, fT) );

	if (hipDeviceSynchronize() != hipSuccess){
		printf("error in synch");
		return -1;	
	}

	//copy massA back in to the host pointers
	START
	for(r = 0, v = 0; r < k1.nreqs; r++){
		gpuErrchk(hipMemcpy(k1.shmid_ptrs[r], &massA_d[v], sizeof(hipfftComplex) * Asize, hipMemcpyDeviceToHost));
		v = v + Asize;	
	}
	FINISH
	printf("%ld,", timediff(sT, fT) );

	gettimeofday(&finish, NULL);
	k1.timespent = timediff(start, finish);

	//free csr from device
	gpuErrchk(hipFree(massA_d));


	//detach from shared mems
	START
	for(r = 0; r < k1.nreqs; r++){
		shmdt((void*) k1.shmid_ptrs[r]);
	}
	FINISH
	printf("%ld,", timediff(sT, fT));

	hipfftDestroy(plancufft);

	send(sockfd, &k1, sizeof(bin), 0);

	printf("%ld\n", k1.timespent);
}



}
